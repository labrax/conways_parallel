#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>

#define SEED

typedef struct _data {
    char * values;
    char * next_values;
    int width;
    int height;
} data;

double rtclock() {
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

__device__ int amount_neighbours(char * source, int x, int y, int width, int height) {
    int i, j;
    int amount = 0;
    for(i = y-1; i <= y+1; i++) {
        for(j = x-1; j <= x+1; j++) {
            //printf("%d %d -- %c\n", j, i, conways_data->values[i*conways_data->width+j]);
            if(i == y && j == x)
                continue;
            if(i >= 0 && i < height
                    && j >= 0 && j < width
                    && source[i*width+j] == '1') {
                amount++;
            }
        }
    }
    assert(amount >= 0 && amount <= 8);
    return amount;
}

__global__ void  operate(char * source, char * goal, int width, int height) {
    int index_i = blockDim.x * blockIdx.x + threadIdx.x;
    int index_j = blockDim.y * blockIdx.y + threadIdx.y;

    int index = index_i*width + index_j;
    if (index_i < height && index_j < width && index < height*width) {
        int amount = amount_neighbours(source, index_j, index_i, width, height);
        if(source[index] == '1') {
            if(amount < 2 || amount > 3)
            goal[index] = '0';
            else
            goal[index] = '1';
        }
        else {
            if(amount == 3)
            goal[index] = '1';
            else
            goal[index] = '0';
        }
            
    }
}

void run_n_times(data * conways_data, int iterations, int number_threads) {
    int i;
    
    char * d_A, * d_B;

    int size = conways_data->height * conways_data->width * sizeof(char);
    hipMalloc((void**) &d_A, size);
    hipMalloc((void**) &d_B, size);
    
    hipMemcpy(d_A, conways_data->values, size, hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(ceil(conways_data->width/(float) threadsPerBlock.x), ceil(conways_data->height/(float) threadsPerBlock.y));
    
    for(i = 0; i < iterations; i++) {
        operate<<<numBlocks, threadsPerBlock>>>(i%2 == 0? d_A : d_B, i%2 == 0? d_B : d_A, conways_data->width, conways_data->height);
        hipDeviceSynchronize();
    }
    
    hipMemcpy(conways_data->values, i%2 == 0? d_A : d_B, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    return;
}

void print_data(data * conways_data) {
    int i, j;
    for(i = 0; i < conways_data->height; i++) {
        for(j = 0; j < conways_data->width; j++) {
            printf("%c ", conways_data->values[i*conways_data->width+j]);
        }
        printf("\n");
    }
    return;
}

int main(void) {
    int w, h, number_threads, seed;
    data conways_data;
    scanf(" %d %d %d", &w, &h, &number_threads);
    conways_data.width = w;
    conways_data.height = h;
    conways_data.values = (char *) malloc(sizeof(char) * w * h);
    conways_data.next_values = (char *) malloc(sizeof(char) * w * h);

    #ifdef SEED
    scanf(" %d", &seed);
    srand(seed);
    #endif
    
    int i, j;
    for(i = 0; i < h; i++) {
        for(j = 0; j < w; j++) {
            #ifdef SEED
            conways_data.values[i * w + j] = '0' + rand() % 2;
            #else
            scanf(" %c", &conways_data.values[i * w + j]);
            #endif
        }
    }

    int iterations;
    scanf(" %d", &iterations);

    double start, end;
    start = rtclock();
    run_n_times(&conways_data, iterations, number_threads);
    end = rtclock();

    print_data(&conways_data);
    printf("%f\n", end-start);

    free(conways_data.values);
    free(conways_data.next_values);
    return 0;
}
